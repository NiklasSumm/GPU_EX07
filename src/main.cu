#include "hip/hip_runtime.h"
/******************************************************************************
 *
 *           XXXII Heidelberg Physics Graduate Days - GPU Computing
 *
 *                 Gruppe : TODO
 *
 *                   File : main.cu
 *
 *                Purpose : n-Body Computation
 *
 ******************************************************************************/

#include <cmath>
#include <ctime>
#include <iostream>
#include <cstdlib>
#include <chCommandLine.h>
#include <chTimer.hpp>
#include <cstdio>
#include <iomanip>

const static int DEFAULT_NUM_ELEMENTS = 1024;
const static int DEFAULT_NUM_ITERATIONS = 5;
const static int DEFAULT_BLOCK_DIM = 128;

const static float TIMESTEP = 1e-6;	  // s
const static float GAMMA = 6.673e-11; // (Nm^2)/(kg^2)

//
// Structures
//
// Here with two AOS (arrays of structures).
//
struct Body_t
{
	float4 posMass;	 /* x = x */
					 /* y = y */
					 /* z = z */
					 /* w = Mass */
	float3 velocity; /* x = v_x*/
					 /* y = v_y */
					 /* z= v_z */

	Body_t() : posMass(make_float4(0, 0, 0, 0)), velocity(make_float3(0, 0, 0)) {}
};

//
// Function Prototypes
//
void printHelp(char *);
void printElement(Body_t *, int, int);

//
// Device Functions
//

//
// Calculate the Distance of two points
//
__device__ float
getDistance(float4 a, float4 b)
{
	float dist_x = b.x - a.x;
	float dist_y = b.y - a.y;
	float dist_z = b.z - a.z;

	return sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
}

//
// Calculate the forces between two bodies
//
__device__ void
bodyBodyInteraction(float4 bodyA, float4 bodyB, float3 &force)
{
	float distance = getDistance(bodyA, bodyB);

	if (distance == 0)
		return;

	float intermediateResult = - GAMMA * (bodyA.w * bodyB.w) / (distance * distance);

	force.x += intermediateResult * ((bodyA.x - bodyB.x) / distance);
	force.y += intermediateResult * ((bodyA.y - bodyB.y) / distance);
	force.z += intermediateResult * ((bodyA.z - bodyB.z) / distance);
}

//
// Calculate the new velocity of one particle
//
__device__ void
calculateSpeed(float mass, float3 &currentSpeed, float3 force)
{
	currentSpeed.x += (force.x / mass) * TIMESTEP;
	currentSpeed.y += (force.y / mass) * TIMESTEP;
	currentSpeed.z += (force.z / mass) * TIMESTEP;
}

//
// n-Body Kernel for the speed calculation
//
__global__ void
simpleNbody_Kernel(int numElements, Body_t *body)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;

	float4 elementPosMass;
	float3 elementForce;
	float3 elementSpeed;

	if (elementId < numElements)
	{
		elementPosMass = body[elementId].posMass;
		elementSpeed = body[elementId].velocity;
		elementForce = make_float3(0, 0, 0);

		for (int i = 0; i < numElements; i++)
		{
			if (i != elementId)
			{
				bodyBodyInteraction(elementPosMass, body[i].posMass, elementForce);
			}
		}

		calculateSpeed(elementPosMass.w, elementSpeed, elementForce);

		body[elementId].velocity = elementSpeed;
	}
}

__global__ void
sharedNbody_Kernel(int numElements, float4 *bodyPos, float3 *bodySpeed)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	int sharedId = threadIdx.x;

	__shared__ float4 sharedBodyPos[1024];

	int tiles = (numElements + 1023) / 1024;

	float4 elementPosMass;
	float3 elementForce;
	float3 elementSpeed;

	if (elementId < numElements)
	{
		elementPosMass = bodyPos[elementId];
		elementSpeed = bodySpeed[elementId];
		elementForce = make_float3(0, 0, 0);

		for (int tile = 0; tile < tiles; tile++)
		{
			__syncthreads();

			if (sharedId < 1024){
				sharedBodyPos[sharedId] = bodyPos[elementId];
			}

			__syncthreads();

			for (int i = 0; i < 1024; i++)
			{
				int id = tile * 1024 + i;
				if (id != elementId && id < numElements)
				{
					bodyBodyInteraction(elementPosMass, sharedBodyPos[i], elementForce);
				}
			}
		}

		calculateSpeed(elementPosMass.w, elementSpeed, elementForce);

		bodySpeed[elementId] = elementSpeed;
	}
}

//
// n-Body Kernel to update the position
// Neended to prevent write-after-read-hazards
//
__global__ void
updatePosition_Kernel(int numElements, Body_t *bodies)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;

	if (elementId < numElements)
	{
		float4 elementPosMass = bodies[elementId].posMass;
		float3 elementSpeed = bodies[elementId].velocity;

		elementPosMass.x += elementSpeed.x * TIMESTEP; 
		elementPosMass.y += elementSpeed.y * TIMESTEP; 
		elementPosMass.z += elementSpeed.z * TIMESTEP; 

		bodies[elementId].posMass = elementPosMass;
	}
}

//
// n-Body Kernel to update the position
// Neended to prevent write-after-read-hazards
//
__global__ void
updatePositionSOA_Kernel(int numElements, float4 *bodyPos, float3 *bodySpeed)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;

	if (elementId < numElements)
	{
		float4 elementPosMass = bodyPos[elementId];
		float3 elementSpeed = bodySpeed[elementId];

		elementPosMass.x += elementSpeed.x * TIMESTEP; 
		elementPosMass.y += elementSpeed.y * TIMESTEP; 
		elementPosMass.z += elementSpeed.z * TIMESTEP; 

		bodyPos[elementId] = elementPosMass;
	}
}

//
// Main
//
int main(int argc, char *argv[])
{
	bool showHelp = chCommandLineGetBool("h", argc, argv);
	if (!showHelp)
	{
		showHelp = chCommandLineGetBool("help", argc, argv);
	}

	if (showHelp)
	{
		printHelp(argv[0]);
		exit(0);
	}

	std::cout << "***" << std::endl
			  << "*** Starting ..." << std::endl
			  << "***" << std::endl;

	ChTimer memCpyH2DTimer, memCpyD2HTimer;
	ChTimer kernelTimer;

	//
	// Allocate Memory
	//
	int numElements = 0;
	chCommandLineGet<int>(&numElements, "s", argc, argv);
	chCommandLineGet<int>(&numElements, "size", argc, argv);
	numElements = numElements != 0 ? numElements : DEFAULT_NUM_ELEMENTS;
	//
	// Host Memory
	//
	bool pinnedMemory = chCommandLineGetBool("p", argc, argv);
	if (!pinnedMemory)
	{
		pinnedMemory = chCommandLineGetBool("pinned-memory", argc, argv);
	}

	bool optimized = chCommandLineGetBool("optimized", argc, argv);

	Body_t *h_particles;
	float4 *h_posMasses;
	float3 *h_speeds;
	if (!pinnedMemory)
	{
		// Pageable
		h_particles = static_cast<Body_t *>(malloc(static_cast<size_t>(numElements * sizeof(*h_particles))));
		h_posMasses = static_cast<float4 *>(malloc(static_cast<size_t>(numElements * sizeof(*h_posMasses))));
		h_speeds = static_cast<float3 *>(malloc(static_cast<size_t>(numElements * sizeof(*h_speeds))));
	}
	else
	{
		// Pinned
		hipHostMalloc(&h_particles, static_cast<size_t>(numElements * sizeof(*h_particles)));
		hipHostMalloc(&h_posMasses, static_cast<size_t>(numElements * sizeof(*h_posMasses)));
		hipHostMalloc(&h_speeds, static_cast<size_t>(numElements * sizeof(*h_speeds)));
	}

	// Init Particles
	//	srand(static_cast<unsigned>(time(0)));
	srand(0); // Always the same random numbers
	for (int i = 0; i < numElements; i++)
	{
		//h_particles[i].posMass.x = 1e-8 * static_cast<float>(rand()); // Modify the random values to
		//h_particles[i].posMass.y = 1e-8 * static_cast<float>(rand()); // increase the position changes
		//h_particles[i].posMass.z = 1e-8 * static_cast<float>(rand()); // and the velocity
		//h_particles[i].posMass.w = 1e4 * static_cast<float>(rand());
		//h_particles[i].velocity.x = 0.0f;
		//h_particles[i].velocity.y = 0.0f;
		//h_particles[i].velocity.z = 0.0f;

		h_particles[i].posMass.x = h_posMasses[i].x = 1e-8 * static_cast<float>(rand()); // Modify the random values to
		h_particles[i].posMass.y = h_posMasses[i].y = 1e-8 * static_cast<float>(rand()); // increase the position changes
		h_particles[i].posMass.z = h_posMasses[i].z = 1e-8 * static_cast<float>(rand()); // and the velocity
		h_particles[i].posMass.w = h_posMasses[i].w = 1e4 * static_cast<float>(rand());
		h_particles[i].velocity.x = h_speeds[i].x = 0.0f;
		h_particles[i].velocity.y = h_speeds[i].y = 0.0f;
		h_particles[i].velocity.z = h_speeds[i].z = 0.0f;
	}

	printElement(h_particles, 0, 0);

	// Device Memory
	Body_t *d_particles;
	float4 *d_posMasses;
	float3 *d_speeds;
	hipMalloc(&d_particles, static_cast<size_t>(numElements * sizeof(*d_particles)));
	hipMalloc(&d_posMasses, static_cast<size_t>(numElements * sizeof(*d_posMasses)));
	hipMalloc(&d_speeds, static_cast<size_t>(numElements * sizeof(*d_speeds)));

	if (h_particles == NULL || d_particles == NULL)
	{
		std::cout << "\033[31m***" << std::endl
				  << "*** Error - Memory allocation failed" << std::endl
				  << "***\033[0m" << std::endl;

		exit(-1);
	}

	//
	// Copy Data to the Device
	//
	memCpyH2DTimer.start();

	if (optimized){
		hipMemcpy(d_posMasses, h_posMasses, static_cast<size_t>(numElements * sizeof(*d_posMasses)), hipMemcpyHostToDevice);
		hipMemcpy(d_speeds, h_speeds, static_cast<size_t>(numElements * sizeof(*d_speeds)), hipMemcpyHostToDevice);
	}
	else{
		hipMemcpy(d_particles, h_particles, static_cast<size_t>(numElements * sizeof(*d_particles)), hipMemcpyHostToDevice);
	}

	memCpyH2DTimer.stop();

	//
	// Get Kernel Launch Parameters
	//
	int blockSize = 0,
		gridSize = 0,
		numIterations = 0;

	// Number of Iterations
	chCommandLineGet<int>(&numIterations, "i", argc, argv);
	chCommandLineGet<int>(&numIterations, "num-iterations", argc, argv);
	numIterations = numIterations != 0 ? numIterations : DEFAULT_NUM_ITERATIONS;

	// Block Dimension / Threads per Block
	chCommandLineGet<int>(&blockSize, "t", argc, argv);
	chCommandLineGet<int>(&blockSize, "threads-per-block", argc, argv);
	blockSize = blockSize != 0 ? blockSize : DEFAULT_BLOCK_DIM;

	if (blockSize > 1024)
	{
		std::cout << "\033[31m***" << std::endl
				  << "*** Error - The number of threads per block is too big" << std::endl
				  << "***\033[0m" << std::endl;

		exit(-1);
	}

	gridSize = ceil(static_cast<float>(numElements) / static_cast<float>(blockSize));

	dim3 grid_dim = dim3(gridSize);
	dim3 block_dim = dim3(blockSize);

	std::cout << "***" << std::endl;
	std::cout << "*** Grid: " << gridSize << std::endl;
	std::cout << "*** Block: " << blockSize << std::endl;
	std::cout << "***" << std::endl;

	bool silent = chCommandLineGetBool("silent", argc, argv);

	updatePosition_Kernel<<<grid_dim, block_dim>>>(numElements, d_particles);

	kernelTimer.start();

	for (int i = 0; i < numIterations; i++)
	{
		if (optimized){
			//sharedNbody_Kernel<<<grid_dim, block_dim>>>(numElements, d_posMasses, d_speeds);
			//updatePositionSOA_Kernel<<<grid_dim, block_dim>>>(numElements, d_posMasses, d_speeds);
		}
		else{
			simpleNbody_Kernel<<<grid_dim, block_dim>>>(numElements, d_particles);
			updatePosition_Kernel<<<grid_dim, block_dim>>>(numElements, d_particles);

			hipMemcpy(h_particles, d_particles, static_cast<size_t>(numElements * sizeof(*h_particles)), hipMemcpyDeviceToHost);
			if (!silent)
			{
				printElement(h_particles, 0, i + 1);
			}
		}
	}

	// Synchronize
	hipDeviceSynchronize();

	// Check for Errors
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
	{
		std::cout << "\033[31m***" << std::endl
				  << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
				  << std::endl
				  << "***\033[0m" << std::endl;

		return -1;
	}

	kernelTimer.stop();

	//
	// Copy Back Data
	//
	memCpyD2HTimer.start();

	//hipMemcpy(h_particles, d_particles, static_cast<size_t>(numElements * sizeof(*d_particles)), hipMemcpyDeviceToHost);
	if (optimized){
		hipMemcpy(h_posMasses, d_posMasses, static_cast<size_t>(numElements * sizeof(*d_posMasses)), hipMemcpyHostToDevice);
		hipMemcpy(h_speeds, d_speeds, static_cast<size_t>(numElements * sizeof(*d_speeds)), hipMemcpyHostToDevice);
	}
	else{
		hipMemcpy(h_particles, d_particles, static_cast<size_t>(numElements * sizeof(*d_particles)), hipMemcpyHostToDevice);
	}

	memCpyD2HTimer.stop();

	// Free Memory
	if (!pinnedMemory)
	{
		free(h_particles);
	}
	else
	{
		hipHostFree(h_particles);
	}

	hipFree(d_particles);

	// Print Meassurement Results
	std::cout << "***" << std::endl
			  << "*** Results:" << std::endl
			  << "***    Num Elements: " << numElements << std::endl
			  << "***    Num Iterations: " << numIterations << std::endl
			  << "***    Threads per block: " << blockSize << std::endl
			  << "***    Time to Copy to Device: " << 1e3 * memCpyH2DTimer.getTime()
			  << " ms" << std::endl
			  << "***    Copy Bandwidth: "
			  << 1e-9 * memCpyH2DTimer.getBandwidth(numElements * sizeof(h_particles))
			  << " GB/s" << std::endl
			  << "***    Time to Copy from Device: " << 1e3 * memCpyD2HTimer.getTime()
			  << " ms" << std::endl
			  << "***    Copy Bandwidth: "
			  << 1e-9 * memCpyD2HTimer.getBandwidth(numElements * sizeof(h_particles))
			  << " GB/s" << std::endl
			  << "***    Time for n-Body Computation: " << 1e3 * kernelTimer.getTime()
			  << " ms" << std::endl
			  << "***" << std::endl;

	return 0;
}

void printHelp(char *argv)
{
	std::cout << "Help:" << std::endl
			  << "  Usage: " << std::endl
			  << "  " << argv << " [-p] [-s <num-elements>] [-t <threads_per_block>]"
			  << std::endl
			  << "" << std::endl
			  << "  -p|--pinned-memory" << std::endl
			  << "    Use pinned Memory instead of pageable memory" << std::endl
			  << "" << std::endl
			  << "  -s <num-elements>|--size <num-elements>" << std::endl
			  << "    Number of elements (particles)" << std::endl
			  << "" << std::endl
			  << "  -i <num-iterations>|--num-iterations <num-iterations>" << std::endl
			  << "    Number of iterations" << std::endl
			  << "" << std::endl
			  << "  -t <threads_per_block>|--threads-per-block <threads_per_block>"
			  << std::endl
			  << "    The number of threads per block" << std::endl
			  << "" << std::endl
			  << "  --silent"
			  << std::endl
			  << "    Suppress print output during iterations (useful for benchmarking)" << std::endl
			  << "" << std::endl;
}

//
// Print one element
//
void printElement(Body_t *particles, int elementId, int iteration)
{
	float4 posMass = particles[elementId].posMass;
	float3 velocity = particles[elementId].velocity;

	std::cout << "***" << std::endl
			  << "*** Printing Element " << elementId << " in iteration " << iteration << std::endl
			  << "***" << std::endl
			  << "*** Position: <"
			  << std::setw(11) << std::setprecision(9) << posMass.x << "|"
			  << std::setw(11) << std::setprecision(9) << posMass.y << "|"
			  << std::setw(11) << std::setprecision(9) << posMass.z << "> [m]" << std::endl
			  << "*** velocity: <"
			  << std::setw(11) << std::setprecision(9) << velocity.x << "|"
			  << std::setw(11) << std::setprecision(9) << velocity.y << "|"
			  << std::setw(11) << std::setprecision(9) << velocity.z << "> [m/s]" << std::endl
			  << "*** Mass: <"
			  << std::setw(11) << std::setprecision(9) << posMass.w << "> [kg]" << std::endl
			  << "***" << std::endl;
}
